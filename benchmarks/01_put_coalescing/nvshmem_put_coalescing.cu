#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "nvshmem.h"

// used to check the status code of cuda routines for errors
#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t _CHECK_result = (stmt);                                              \
        if (hipSuccess != _CHECK_result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(_CHECK_result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

// used to check the status code of NVSHMEM routines for errors
#define NVSHMEM_CHECK(stmt)                                                                \
    do {                                                                                   \
        int _CHECK_result = (stmt);                                                               \
        if (NVSHMEMX_SUCCESS != _CHECK_result) {                                                  \
            fprintf(stderr, "[%s:%d] nvshmem failed with error %d \n", __FILE__, __LINE__, \
                    _CHECK_result);                                                               \
            exit(-1);                                                                      \
        }                                                                                  \
    } while (0)

constexpr size_t N_ELEMS{1024 * 1024};
constexpr int TEST_1_SEND_DONE{1};
constexpr int TEST_2_SEND_DONE{2};

__global__ void exchange_data(int this_pe,
                              uint8_t *const data,
                              int *const flag) {
    const int other_pe = static_cast<int>(!this_pe); // there are two PEs in total

    // we only use one thread
    if (threadIdx.x != 0) {
        return;
    }

    // PE 0 is the sender
    if (this_pe == 0) {
        // populate data to send to PE 1
        for (size_t i{0}; i < N_ELEMS; ++i) {
            // write lower bits of index to every element
            data[i] = static_cast<uint8_t >(i);
        }

        // sync with other PE to make them start simultaneously
        nvshmem_barrier_all();

        long long start_time = clock64();

        // send data to other PE at same position
        for (size_t i{0}; i < N_ELEMS; ++i) {
            nvshmem_uint8_put_nbi(data + i,
                                  data + i,
                                  1,
                                  other_pe);
        }

        // let following mem operations be executed after the previous sending
        nvshmem_fence();

        // atomically set memory flag at other PE to signal that all previous send operation must have been completed (see fence)
        nvshmem_int_atomic_set(flag, TEST_1_SEND_DONE, 1);

        // make sure all send buffers are reusable
        nvshmem_quiet();

        long long stop_time = clock64();
        long long elapsed_time = stop_time - start_time;

        printf("Sender: time for sending %lu elems separately and calling nvshmem_quiet: %lld (clock start %lld, clock stop %lld)\n", N_ELEMS, elapsed_time, start_time, stop_time);

        // TODO: return result in CSV format

        // synchronize with other PE to make them start next test simultaneously
        nvshmem_barrier_all();

        start_time = clock64();

        // send data in one go
        nvshmem_uint8_put_nbi(data,
                              data,
                              N_ELEMS,
                              other_pe);

        // let following mem operations be executed after the previous sending
        nvshmem_fence();

        // atomically set memory flag at other PE to signal that all previous send operation must have been completed (see fence)
        nvshmem_int_atomic_set(flag, TEST_2_SEND_DONE, 1);

        // make sure all send buffers are reusable
        nvshmem_quiet();

        stop_time = clock64();
        elapsed_time = stop_time - start_time;

        printf("Sender: time for sending %lu elems at once and calling nvshmem_quiet: %lld (clock start %lld, clock stop %lld)\n", N_ELEMS, elapsed_time, start_time, stop_time);

    } else { // PE 1 is the receiver
        // sync with other PE to make them start simultaneously
        nvshmem_barrier_all();

        long long start_time = clock64();

        // wait until flag has been delivered, this then indicates all previous data has been delivered
        nvshmemi_wait_until(flag, NVSHMEM_CMP_EQ, TEST_1_SEND_DONE);

        long long stop_time = clock64();
        auto elapsed_time = stop_time - start_time;

        printf("Receiver: time until all %lu elems have been received separately: %lld (clock start %lld, clock stop %lld)\n", N_ELEMS, elapsed_time, start_time, stop_time);

        // verify correctness
        for (size_t i{0}; i < N_ELEMS; ++i) {
            // write lower bits of index to every element
            assert(data[i] == static_cast<uint8_t>(i));
        }

        // TODO: emit time in proper CSV format

        // reset receive buffer for next test
        memset(data, 0, N_ELEMS);

        // synchronize with other PE to make them start next test simultaneously
        nvshmem_barrier_all();

        start_time = clock64();

        // wait until flag has been delivered, this then indicates all previous data has been delivered
        nvshmemi_wait_until(flag, NVSHMEM_CMP_EQ, TEST_2_SEND_DONE);

        stop_time = clock64();
        elapsed_time = stop_time - start_time;

        printf("Receiver: time until all %lu elems have been received at once: %lld (clock start %lld, clock stop %lld)\n", N_ELEMS, elapsed_time, start_time, stop_time);

        // verify correctness
        for (size_t i{0}; i < N_ELEMS; ++i) {
            // write lower bits of index to every element
            assert(data[i] == static_cast<uint8_t>(i));
        }

        // TODO: emit time in proper CSV format

    }
}

int main(int argc, char *argv[]) {
    // init nvshmem
    int n_pes, this_pe;
    hipStream_t stream;

    nvshmem_init();
    this_pe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    n_pes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    printf("Hello from PE %d of %d\n", this_pe, n_pes);
    hipSetDevice(this_pe);
    hipStreamCreate(&stream);

    // this test is supposed to be executed on 2 PEs, each sends and receives data from the other PE
    assert(n_pes == 2);

    // allocate symmetric device memory for sending/receiving the data
    auto *const data = static_cast<uint8_t *>(nvshmem_malloc(N_ELEMS));
    auto *const flag = static_cast<int *>(nvshmem_malloc(sizeof(int)));

    // call benchmarking kernel
    void *args[] = {&this_pe,
                    const_cast<uint8_t **>(&data),
                    const_cast<int **>(&flag)};
    NVSHMEM_CHECK(nvshmemx_collective_launch((const void *) exchange_data, 1, 1, args, 1024 * 4, stream));

    // wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());


    // TODO: print results in suitable CSV format

//    std::ofstream outfile;
//    outfile.open("results.csv");
//    outfile << "type, node_count,in n,out n" << std::endl;
//
//    outfile.close();
}
