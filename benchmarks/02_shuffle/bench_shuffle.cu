#include <iostream>
#include "shuffle.h"

struct shuffle_tuple {
    uint64_t id;
    uint64_t data[7];
};

struct create_tuple_result {
    shuffle_tuple **tuples;
    uint64_t *num_tuples;
};

// configuration for this shuffle example
constexpr uint8_t KEY_OFFSET = 0; // key is first item in shuffle_tuple

// creates local tuples in device memory
shuffle_tuple *create_tuples(uint64_t *tuple_ids, size_t num_tuples) {
    size_t localMemSize = num_tuples * sizeof(shuffle_tuple);
    // allocate memory for tuples on host
    auto *localTuplesCPU = static_cast<shuffle_tuple *>(malloc(localMemSize));

    // fill in ids of the tuples as ascending integers with an offset depending on the PE_id
    for (size_t i{0}; i < num_tuples; ++i) {
        localTuplesCPU[i].id = tuple_ids[i];
    }

    // allocate device memory for the local tuples
    shuffle_tuple *localTuplesGPU;
    CUDA_CHECK(hipMalloc(&localTuplesGPU, num_tuples * sizeof(shuffle_tuple)));

    // copy tuples to device memory
    CUDA_CHECK(hipMemcpy(localTuplesGPU, localTuplesCPU, localMemSize, hipMemcpyHostToDevice));

    // free CPU memory
    free(localTuplesCPU);

    return localTuplesGPU;
}

create_tuple_result create_all_local_tuples(int table_size) {
    int nPes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    shuffle_tuple **tuples = (shuffle_tuple **) malloc(nPes * sizeof(shuffle_tuple *));
    uint64_t *num_tuples = (uint64_t *) malloc(nPes * sizeof(uint64_t));

    for (int i = 0; i < nPes; ++i) {
        const uint64_t count_tuples = table_size / nPes;  // distribute the table_size evenly across all PEs
        const auto tuple_ids = new uint64_t[count_tuples];
        for (uint64_t j = 0; j < count_tuples; ++j) {
            tuple_ids[j] = j + count_tuples * i;  // adjusted to ensure unique ids
        }
        printf("PE %d has tuple ids: ", i);
        for (uint64_t j = 0; j < count_tuples; ++j) {
            printf("%lu ", tuple_ids[j]);
        }
        printf("\n");
        tuples[i] = create_tuples(tuple_ids, count_tuples);
        num_tuples[i] = count_tuples;
    }
    // print num tuples for all pes
    for (int i = 0; i < nPes; ++i) {
        printf("PE %d has %lu tuples\n", i, num_tuples[i]);
    }
    return create_tuple_result{
        tuples,
        num_tuples
    };
}

void call_shuffle(hipStream_t &stream, shuffle_tuple **local_tuples, uint64_t *num_tuples) {

    int thisPe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    int nPes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);

    // shuffle data
    const ShuffleResult result =
        shuffle(reinterpret_cast<const uint8_t *>(local_tuples[thisPe]), sizeof(shuffle_tuple), num_tuples[thisPe],
                KEY_OFFSET, stream, NVSHMEM_TEAM_WORLD);

//    // check that the local result contains the correct tuples
//    for (uint64_t i{0}; i < result.partitionSize; ++i) {
//        // modulo of received tuples should be this PE's ID
//        assert(reinterpret_cast<uint64_t *>(result.tuples)[i * 8] % nPes == thisPe);
//    }
}

int main(int argc, char *argv[]) {
    // Check if a table size argument is given
    if (argc < 2) {
        std::cout << "Usage: " << argv[0] << " <table_size>" << std::endl;
        return 1;
    }

    // Convert argument to integer
    int table_size = std::stoi(argv[1]);

    int nPes, thisPe;
    hipStream_t stream;

    nvshmem_init();
    thisPe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    printf("PE %d: table size %d\n", thisPe, table_size);
    hipStreamCreate(&stream);

    // Pass the table size to the tuple creation function
    const create_tuple_result tuple_result = create_all_local_tuples(table_size);
    call_shuffle(stream, tuple_result.tuples, tuple_result.num_tuples);

    nvshmem_finalize();
    return 0;
}

