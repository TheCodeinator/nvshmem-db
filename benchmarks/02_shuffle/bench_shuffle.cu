#include <iostream>
#include <hip/hip_runtime.h>
#include "nvshmem.h"

int main(int argc, char *argv[]) {
    // Check if a table size argument is given
    if (argc < 2) {
        std::cout << "Usage: " << argv[0] << " <table_size>" << std::endl;
        return 1;
    }

    // Convert argument to integer
    int table_size = std::stoi(argv[1]);

    int nPes, thisPe;
    hipStream_t stream;

    nvshmem_init();
    thisPe = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(thisPe);
    hipStreamCreate(&stream);

    printf("PE %d: table size %d\n", thisPe, table_size);

    nvshmem_finalize();
    return 0;
}
