#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <array>
#include <vector>
#include "nvshmem.h"
#include "NVSHMEMUtils.cuh"
#include "Macros.cuh"

constexpr long long MAX_SEND_SIZE{1024 * 1024};

// TODO: verify results make sense and benchmark code is bug-free

// from 2 go up to the max packet size in exponential steps
constexpr size_t N_TESTS{log2const(MAX_SEND_SIZE) + 1};

__global__ void exchange_data(uint8_t *const data_src,
                              uint8_t *const data_dest,
                              const uint64_t n_bytes,
                              const uint32_t msg_size) {

    // send number of bytes in total, split up in batches of given message size
    for (size_t i{0}; i < (n_bytes / msg_size); ++i) {
        nvshmem_uint8_fcollect(NVSHMEM_TEAM_WORLD, data_dest, data_src, msg_size);
    }

    // sync all PEs
    nvshmem_barrier_all();
}

/**
 * cmd arguments:
 * 0) program name (implicit)
 * 1) number of bytes to send per PE
 */
int main(int argc, char *argv[]) {
    // init nvshmem
    int n_pes, this_pe;
    hipStream_t stream;

    assert(argc == 2);
    const u_int64_t n_bytes = std::stoull(argv[1]);
    constexpr u_int32_t grid_dim = 1;
    constexpr u_int32_t block_dim = 1;

    nvshmem_init();
    this_pe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    n_pes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    hipSetDevice(this_pe);
    hipStreamCreate(&stream);

    if (n_pes != 2) {
        throw std::logic_error(
                "this test is supposed to be executed on 2 PEs, each sends and receives data from the other PE.");
    }

    if (n_bytes / MAX_SEND_SIZE < 1 || n_bytes % MAX_SEND_SIZE != 0) {
        throw std::logic_error("Make sure that the number of bytes to send is divisible by the maximum send size (" +
                               std::to_string(MAX_SEND_SIZE) + ").");
    }

    // allocate symmetric device memory for sending/receiving the data
    auto *const data_src = static_cast<uint8_t *>(nvshmem_malloc(MAX_SEND_SIZE));
    // dest array has space for each PE's data
    auto *const data_dest = static_cast<uint8_t *>(nvshmem_malloc(MAX_SEND_SIZE * n_pes));

    std::vector<std::pair<uint32_t, std::chrono::nanoseconds>> measurements{};
    measurements.reserve(N_TESTS);

    for (size_t test{0}; test < N_TESTS; ++test) {
        const uint32_t msg_size = int_pow(2, test);
        measurements.emplace_back(msg_size,
                                  time_kernel(exchange_data, grid_dim, block_dim, 1024 * 4, stream,
                                              data_src, data_dest, n_bytes, msg_size));
    }

    // deallocate all the memory that has been allocated
    nvshmem_free(data_src);
    nvshmem_free(data_dest);

    if (this_pe == 0) {
        for (const auto &meas: measurements) {
            std::cout << "msg_size = " << meas.first << ", throughput = " << gb_per_sec(meas.second, n_bytes) << " GB/s"
                      << std::endl;
        }
    }

    // TODO: print results in suitable CSV format

//    std::ofstream outfile;
//    outfile.open("results.csv");
//    outfile << "type, node_count,in n,out n" << std::endl;
//
//    outfile.close();
}
