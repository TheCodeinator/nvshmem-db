#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <vector>
#include <bit>
#include "nvshmem.h"
#include "NVSHMEMUtils.cuh"


__global__ void generalized_benchmark(uint8_t *data_source,
                                      uint8_t *data_sink,
                                      const uint64_t num_bytes,
                                      const uint64_t message_size,
                                      const uint64_t buffer_size) {
    const int this_pe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    const int n_pes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    const uint64_t thread_id = global_thread_id();
    const uint64_t thread_count = global_thread_count();

    if (this_pe != 0 && thread_id == 0) {
        // wait to receive all data from PE 0
        nvshmem_barrier_all();
        return; // nothing else to do for the receiver
    }

    const uint64_t n_messages_per_buffer = buffer_size / message_size;

    for (uint64_t i = 0; i < num_bytes / (message_size * thread_count); ++i) {
        // each thread sends from a different source pointer in each iteration
        const uint64_t thread_offset = ((thread_id + i) % n_messages_per_buffer) * message_size;
        // each thread sends to a different destination PE and we rotate destinations between threads each iteration
        const uint32_t thread_destination_pe = (thread_id + i) % n_pes;

        nvshmem_uint8_put_nbi(
                data_sink + thread_offset,
                data_source + thread_offset,
                message_size,
                thread_destination_pe);
    }

    if (thread_id == 0) {
        nvshmem_quiet();
        // notifiy PEs > 0 that we (PE 0) have finished sending all data and sync with them
        nvshmem_barrier_all();
    }
}

// Do a barrier operation to prevent compile from optimizing out empty kernel
__global__ void warmup() {

    const uint32_t thread_global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_global_id == 0) {
        nvshmem_barrier_all();
    }

}

int main(int argc, char *argv[]) {
    if (argc != 6 && argc != 7) {
        throw std::invalid_argument(
                "Usage: " + std::string(argv[0]) +
                " <grid_dims> <block_dims> <num_hosts> <num_bytes> <max_send_size> [<min_send_size>]");
    }

    hipStream_t stream;

    const uint32_t grid_dim = std::stoi(argv[1]);
    const uint32_t block_dim = std::stoi(argv[2]);
    const uint32_t num_hosts = std::stoi(argv[3]);

    // the number of bytes that are sent in total per kernel
    const uint64_t num_bytes = std::stoi(argv[4]);

    // the maximum number of bytes that are sent with a single nvshmem put call (increases in powers of 2 starting at 1)
    const uint64_t max_send_size = std::stoi(argv[5]);

    const uint64_t min_send_size = argc == 7 ? std::stoi(argv[6]) : 1;

    if (min_send_size > max_send_size) {
        throw std::invalid_argument("min_send_size must be smaller than max_send_size");
    }

    const uint64_t buffer_size = grid_dim * block_dim * max_send_size;

    if (std::popcount(max_send_size) != 1) {
        throw std::invalid_argument("max_send_size must be a power of 2");
    }

    if (num_bytes / (buffer_size) < 1) {
        throw std::invalid_argument(
                "num_bytes must be greater than grid_dim * block_dim * max_send_size (= " +
                std::to_string(buffer_size) + ")");
    }

    if (num_bytes % (buffer_size) != 0) {
        throw std::invalid_argument(
                "num_bytes must be a multiple of grid_dim * block_dim * max_send_size (= " +
                std::to_string(buffer_size) + ")");
    }

    nvshmem_init();
    const int32_t this_pe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    const int32_t n_pes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    hipSetDevice(this_pe);
    hipStreamCreate(&stream);

    if (n_pes < 2) {
        throw std::logic_error(
                "This test has to be started with 2 PEs or more. PE 0 is the sender, all others are receivers.");
    }

    // each thread is allocated one MAX_SEND_SIZE element, which is re-sent until num_bytes are sent
    auto *data_source = static_cast<uint8_t *>(nvshmem_malloc(buffer_size));
    auto *data_sink = static_cast<uint8_t *>(nvshmem_malloc(buffer_size));

    // warm up device
    collective_launch(warmup, grid_dim, block_dim, 0, stream);

    for (uint64_t message_size = min_send_size; message_size <= max_send_size; message_size <<= 1) {
        const auto time_taken = time_kernel(generalized_benchmark, grid_dim, block_dim, 0, stream,
                                            data_source, data_sink, num_bytes, message_size, max_send_size, buffer_size);
        if (this_pe == 0) {
            std::cout << message_size
                      << " bytes sized packages took " << time_taken.count()
                      << " nanoseconds "
                      << "(" << gb_per_sec(time_taken, num_bytes) << " GB/s)"
                      << std::endl;
        }
    }

    nvshmem_free(data_source);
    nvshmem_free(data_sink);
    nvshmem_finalize();
    return 0;
}
