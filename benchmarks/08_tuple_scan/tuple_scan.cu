#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <unistd.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <type_traits>
#include <vector>
#include "nvshmem.h"
#include "NVSHMEMUtils.cuh"
#include "shuffle.h"


template<typename Tuple>
__global__ void build_thread_offsets(const ShuffleData<Tuple> *data, ThreadOffsets<Tuple> *offsets) {
    const uint32_t thread_id = global_thread_id();

    for (uint32_t i = thread_id; i < data->tuple_count; i += data->thread_count) {
        const Tuple &tuple = data->device_tuples[i];
        const uint32_t dest = distribute(tuple.key, data->pe_count);

        // increment the count for this thread for the current batch and destination (translate it to the offset later)
        ++(*offsets->getOffset(i / data->send_buffer_size_in_tuples, thread_id, dest));
    }

    for (uint32_t i = thread_id; i < data->batch_count * data->pe_count; i += data->thread_count) {
        uint32_t batch = i / data->pe_count;
        uint32_t dest = i % data->pe_count;
        uint32_t currentOffset = 0;
        for (uint32_t thread = 0; thread < data->thread_count; ++thread) {
            uint32_t *offset = offsets->getOffset(batch, thread, dest);
            uint32_t tmp = *offset;
            *offset = currentOffset;
            currentOffset += tmp;
        }
    }
}

template<typename Tuple>
__global__ void generate_tuples(uint64_t tuple_size, uint64_t tuple_count, Tuple *tuples) {
    const uint32_t thread_id = global_thread_id();
    const uint32_t thread_count = global_thread_count();

    for(decltype(Tuple::key) i = thread_id; i < tuple_count; i += thread_count) {
        tuples[i].key = i;
    }
}

template<OffsetMode offset_mode, typename Tuple>
__global__ void tuple_scan(ShuffleData<Tuple> *data, SendBuffers<Tuple> *buffers, ThreadOffsets<Tuple> *offsets) {
    if(data == nullptr || buffers == nullptr || offsets == nullptr)
        return;

    const uint32_t thread_id = global_thread_id();

    const uint iteration_to_send = data->send_buffer_size_in_tuples / data->thread_count;
    uint iteration = 0;

    const uint max_index = data->thread_count * static_cast<uint32_t>(ceil(static_cast<double>(data->tuple_count) / data->thread_count));
    for(uint64_t i = thread_id; i < max_index; i += data->thread_count) {
        if (i < data->tuple_count) {
            const Tuple &tuple = data->device_tuples[i];
            const uint dest = distribute(tuple.key, data->pe_count);

            uint32_t offset;
            if constexpr(offset_mode == OffsetMode::SYNC_FREE) {
                auto thread_offset = offsets->getOffset(i / data->send_buffer_size_in_tuples, thread_id, dest);
                offset = *thread_offset;
                *thread_offset += 1;
            } else if constexpr(offset_mode == OffsetMode::ATOMIC_INCREMENT) {
                // increment the offset for this destination atomically (atomicAdd returns the value before increment)
                offset = atomicAdd(buffers->currentOffsets() + dest, 1);
            } else {
                assert(false);
            }

            buffers->currentBuffer()[dest * data->send_buffer_size_in_tuples + offset] = tuple;
            //memcpy(buffers->currentBuffer() + dest * data->send_buffer_size_in_bytes +
            //       offset * data->tuple_size, // to dest-th buffer with offset position
            //       tuple,
            //       data->tuple_size);

        }

        if(++iteration % iteration_to_send == 0 || i + (data->thread_count - thread_id) >= data->tuple_count) {
            __syncthreads(); // sync threads before send operation (to ensure that all threads have written their data into the buffer)
            if(thread_id == 0) {
                buffers->useNextBuffer(); // switch to the next buffer
                buffers->resetBuffer(buffers->currentBufferIndex()); // reset the offsets of the current buffer
            }
            __syncthreads(); // sync threads after send operation
        }
    }
}

struct BenchmarkArgs {
    uint32_t grid_dim_min;
    uint32_t grid_dim_step;
    uint32_t grid_dim_max;

    uint32_t block_dim_min;
    uint32_t block_dim_step;
    uint32_t block_dim_max;

    uint32_t send_buffer_size_multiplier_min;
    uint32_t send_buffer_size_multiplier_step;
    uint32_t send_buffer_size_multiplier_max;

    uint64_t tuple_count;

    uint32_t shared_mem;
};

template<OffsetMode offset_mode, uint16_t tuple_size>
void benchmark(BenchmarkArgs args) {
    typedef Tuple<uint64_t, uint8_t[tuple_size - sizeof(uint64_t)]> TupleType;

    assert(tuple_size == sizeof(TupleType));

    for(uint32_t grid_dim_tmp = args.grid_dim_min; grid_dim_tmp <= args.grid_dim_max; grid_dim_tmp += args.grid_dim_step) {
        const auto grid_dim = std::max<uint32_t>(grid_dim_tmp, 1);
        for(uint32_t block_dim_tmp = args.block_dim_min; block_dim_tmp <= args.block_dim_max; block_dim_tmp += args.block_dim_step) {
            const auto block_dim = std::max<uint32_t>(block_dim_tmp, 1);
            for(uint32_t send_buffer_size_multiplier_tmp = args.send_buffer_size_multiplier_min;
                send_buffer_size_multiplier_tmp <= args.send_buffer_size_multiplier_max;
                send_buffer_size_multiplier_tmp += args.send_buffer_size_multiplier_step)
            {
                hipSetDevice(nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE));

                hipStream_t stream;
                hipStreamCreate(&stream);

                TupleType *tuples;
                uint64_t tuples_size = tuple_size * args.tuple_count;
                CUDA_CHECK(hipMalloc(&tuples, tuples_size));

                generate_tuples<<<args.grid_dim_max, args.block_dim_max, args.shared_mem, stream>>>(tuple_size, args.tuple_count, tuples);
                hipStreamSynchronize(stream);

                time_kernel(tuple_scan<offset_mode, TupleType>, 1, 1, args.shared_mem, stream, nullptr, nullptr, nullptr);

                const auto send_buffer_size_multiplier = std::max<uint32_t>(send_buffer_size_multiplier_tmp, 1);
                ShuffleData data(tuples,
                                 nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD),
                                 grid_dim,
                                 block_dim,
                                 args.tuple_count,
                                 send_buffer_size_multiplier,
                                 true);
                SendBuffers send_buffers(&data);
                ThreadOffsets thread_offsets(&data);

                auto build_thread_offsets_time = std::chrono::nanoseconds(0);
                if constexpr(offset_mode == OffsetMode::SYNC_FREE) {
                    build_thread_offsets_time = time_kernel(
                            build_thread_offsets<TupleType>,
                            grid_dim, block_dim, args.shared_mem, stream, data.device_data,
                            thread_offsets.device_offsets);
                }

                const auto scan_time = time_kernel(
                        tuple_scan<offset_mode, TupleType>,
                        grid_dim, block_dim, args.shared_mem, stream,
                        data.device_data, send_buffers.device_buffers,
                        thread_offsets.device_offsets);

                const auto time_taken = std::chrono::duration_cast<std::chrono::nanoseconds>(scan_time + build_thread_offsets_time).count();

                if(hipGetLastError() == hipSuccess) {
                    std::cout << "08_tuple_scan" << ","
                              << args.tuple_count << ","
                              << tuple_size << ","
                              << grid_dim << ","
                              << block_dim << ","
                              << send_buffer_size_multiplier << ","
                              << static_cast<int>(offset_mode) << ","
                              << time_taken << ","
                              << gb_per_sec(scan_time + build_thread_offsets_time, tuples_size) << std::endl;
                }

                hipFree(tuples);
                hipStreamDestroy(stream);
            }
        }
    }
}

/**
 * Arguments:
 * 0. program path (implicit)
 * 1. grid_dim_min
 * 2. grid_dim_step
 * 3. grid_dim_iterations
 * 4. block_dim_min
 * 5. block_dim_step
 * 6. block_dim_iterations
 * 7. send_buffer_size_multiplier_min
 * 8. send_buffer_size_multiplier_step
 * 9. send_buffer_size_multiplier_iterations
 * 10. tuple_count
 */
int main(int argc, char *argv[]) {
    if (argc != 11) {
        std::cerr << "Usage: "
            << argv[0]
            << " <grid_dim_min> <grid_dim_step> <grid_dim_iterations>"
            << " <block_dim_min> <block_dim_step> <block_dim_iterations>"
            << " <send_buffer_size_multiplier_min> <send_buffer_size_multiplier_step> <send_buffer_size_multiplier_iterations>"
            << " <tuple_count>"
            << std::endl;
        return 1;
    }

    const uint32_t grid_dim_min = std::stoul(argv[1]);
    const uint32_t grid_dim_step = std::stoul(argv[2]);
    const uint32_t grid_dim_iterations = std::stoul(argv[3]);
    assert(grid_dim_step > 0);
    assert(grid_dim_iterations > 0);

    const uint32_t block_dim_min = std::stoul(argv[4]);
    const uint32_t block_dim_step = std::stoul(argv[5]);
    const uint32_t block_dim_iterations = std::stoul(argv[6]);
    assert(block_dim_step > 0);
    assert(block_dim_iterations > 0);

    const uint32_t send_buffer_size_multiplier_min = std::stoul(argv[7]);
    const uint32_t send_buffer_size_multiplier_step = std::stoul(argv[8]);
    const uint32_t send_buffer_size_multiplier_iterations = std::stoul(argv[9]);
    assert(send_buffer_size_multiplier_step > 0);
    assert(send_buffer_size_multiplier_iterations > 0);

    const uint64_t tuple_count = std::stoull(argv[10]);

    const uint32_t grid_dim_max = grid_dim_min + grid_dim_step * (grid_dim_iterations - 1);
    const uint32_t block_dim_max = block_dim_min + block_dim_step * (block_dim_iterations - 1);
    const uint32_t send_buffer_size_multiplier_max = send_buffer_size_multiplier_min + send_buffer_size_multiplier_step * (send_buffer_size_multiplier_iterations - 1);

    assert(block_dim_max <= 1024);
    uint32_t shared_mem = 4096;

    nvshmem_init();

    std::cout << "type,tuple_count,tuple_size,grid_dim,block_dim,send_buffer_size_multiplier,offset_mode,scan_time_nanoseconds,throughput_gb_s" << std::endl;
    BenchmarkArgs benchmarkArgs(grid_dim_min, grid_dim_step, grid_dim_max,
                                   block_dim_min, block_dim_step, block_dim_max,
                                   send_buffer_size_multiplier_min, send_buffer_size_multiplier_step, send_buffer_size_multiplier_max,
                                   tuple_count,
                                   shared_mem);

    benchmark<OffsetMode::SYNC_FREE, 32>(benchmarkArgs);
    benchmark<OffsetMode::SYNC_FREE, 64>(benchmarkArgs);
    benchmark<OffsetMode::SYNC_FREE, 128>(benchmarkArgs);
    benchmark<OffsetMode::SYNC_FREE, 256>(benchmarkArgs);
    benchmark<OffsetMode::SYNC_FREE, 512>(benchmarkArgs);
    benchmark<OffsetMode::SYNC_FREE, 1024>(benchmarkArgs);

    benchmark<OffsetMode::ATOMIC_INCREMENT, 32>(benchmarkArgs);
    benchmark<OffsetMode::ATOMIC_INCREMENT, 64>(benchmarkArgs);
    benchmark<OffsetMode::ATOMIC_INCREMENT, 128>(benchmarkArgs);
    benchmark<OffsetMode::ATOMIC_INCREMENT, 256>(benchmarkArgs);
    benchmark<OffsetMode::ATOMIC_INCREMENT, 512>(benchmarkArgs);
    benchmark<OffsetMode::ATOMIC_INCREMENT, 1024>(benchmarkArgs);

    nvshmem_finalize();
    return 0;
}
