#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "nvshmem.h"
#include "nvshmemx.h"

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    nvshmem_int_p(destination, mype, peer);
}

int check(int msg) {
    const int thisPe = nvshmem_my_pe();
    const int nPes = nvshmem_n_pes();

    int ringPredecessor = thisPe == 0 ? nPes - 1 : thisPe - 1;
    assert(msg == ringPredecessor);
}

int main(void) {
    int mype_node, msg;
    hipStream_t stream;

    nvshmem_init();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipSetDevice(mype_node);
    hipStreamCreate(&stream);

    int *destination = (int *) nvshmem_malloc(sizeof(int));

    simple_shift<<<1, 1, 0, stream>>>(destination);
    nvshmemx_barrier_all_on_stream(stream);
    hipMemcpyAsync(&msg, destination, sizeof(int), hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    check(msg);
    printf("%d: received message %d\n", nvshmem_my_pe(), msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    return 0;
}
