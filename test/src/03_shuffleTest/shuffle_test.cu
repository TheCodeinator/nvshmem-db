#include "hip/hip_runtime.h"
#include <iostream>
#include "shuffle.h"

struct shuffle_tuple {
    uint64_t id;
    uint64_t data[7];
};

struct create_tuple_result {
    shuffle_tuple **tuples;
    uint64_t *num_tuples;
};

// configuration for this shuffle example
constexpr uint8_t KEY_OFFSET = 0; // key is first item in shuffle_tuple

// creates local tuples in device memory
shuffle_tuple *create_tuples(uint64_t *tuple_ids, size_t num_tuples) {
    size_t localMemSize = num_tuples * sizeof(shuffle_tuple);
    // allocate memory for tuples on host
    auto *localTuplesCPU = static_cast<shuffle_tuple *>(malloc(localMemSize));

    // fill in ids of the tuples as ascending integers with an offset depending on the PE_id
    for (size_t i{0}; i < num_tuples; ++i) {
        localTuplesCPU[i].id = tuple_ids[i];
    }

    // allocate device memory for the local tuples
    shuffle_tuple *localTuplesGPU;
    CUDA_CHECK(hipMalloc(&localTuplesGPU, num_tuples * sizeof(shuffle_tuple)));

    // copy tuples to device memory
    CUDA_CHECK(hipMemcpy(localTuplesGPU, localTuplesCPU, localMemSize, hipMemcpyHostToDevice));

    // free CPU memory
    free(localTuplesCPU);

    return localTuplesGPU;
}

create_tuple_result create_all_local_tuples() {
    int nPes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);
    shuffle_tuple **tuples = (shuffle_tuple **) malloc(nPes * sizeof(shuffle_tuple *));
    uint64_t *num_tuples = (uint64_t *) malloc(nPes * sizeof(uint64_t));
    for (int i = 0; i < nPes; ++i) {
        const uint64_t count_tuples = 5 + 3 * i;
        const auto tuple_ids = new uint64_t[count_tuples];
        for (int j = 0; j < count_tuples; ++j) {
            tuple_ids[j] = j + 5 * i;
        }
        printf("PE %d has tuple ids: ", i);
        for (int j = 0; j < count_tuples; ++j) {
            printf("%lu ", tuple_ids[j]);
        }
        printf("\n");
        tuples[i] = create_tuples(tuple_ids, count_tuples);
        num_tuples[i] = count_tuples;
    }
    // print num tuples for all pes
    for (int i = 0; i < nPes; ++i) {
        printf("PE %d has %lu tuples\n", i, num_tuples[i]);
    }
    return create_tuple_result{
            tuples,
            num_tuples
    };
}

__global__ void printGPUTuples(shuffle_tuple *tuples, uint64_t numTuples, int thisPe) {
    if (threadIdx.x == 0) {
        printf("GPU PE %d start tuples: ", thisPe);
        for (uint64_t i{0}; i < numTuples; ++i) {
            printf("%lu ", tuples[i].id);
        }
        printf("\n");
    }
}

// before shuffle::
// PE 0: 0 1 2 3 4
// PE 1: 5 6 7 8 9 10 11 12

// after shuffle:
// PE 0: 0 2 4 6 8 10 12
// PE 1: 1 3 5 7 9 11
void call_shuffle(hipStream_t &stream, shuffle_tuple **local_tuples, uint64_t *num_tuples) {

    int thisPe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
    int nPes = nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD);

    printGPUTuples<<<1, 1, 0, stream>>>(local_tuples[thisPe], num_tuples[thisPe], thisPe);

    // shuffle data
    const ShuffleResult result =
            shuffle(reinterpret_cast<const uint8_t *>(local_tuples[thisPe]), sizeof(shuffle_tuple), num_tuples[thisPe],
                    KEY_OFFSET, stream, NVSHMEM_TEAM_WORLD);

    // check that the local result contains the correct tuples
    for (uint64_t i{0}; i < result.partitionSize; ++i) {
        // modulo of received tuples should be this PE's ID
        assert(reinterpret_cast<uint64_t *>(result.tuples)[i * 8] % nPes == thisPe);
    }
}

int main() {
    int nPes, thisPe;
    hipStream_t stream;

    nvshmem_init();
    thisPe = nvshmem_team_my_pe(NVSHMEM_TEAM_WORLD);
//    hipSetDevice(thisPe);
    hipStreamCreate(&stream);

    const create_tuple_result tuple_result = create_all_local_tuples();
    call_shuffle(stream, tuple_result.tuples, tuple_result.num_tuples);

    nvshmem_finalize();
    return 0;
}
